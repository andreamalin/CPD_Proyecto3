#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <string.h>
#include "CImg.h"
using namespace cimg_library;

/*
  M_PI hace referencia al valor numérico Pi
  DEG2RAG es la constante para convertir grados a radianes
  degreeBins representa la cantidad de bins en los que se dividen
  180 grados en el acumulador utilizado en la transformada de Hough
  radInc representa el incremento en radianes
*/
#define M_PI 3.14159265358979323846
#define DEG2RAD (M_PI/180.0f)
const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const float radInc = degreeInc * DEG2RAD;
const int rBins = 100;

//*****************************************************************
/*
  Transformada de Hough en el CPu.
  Se identifican los pixeles casi blancos y se actualiza el
  acumulador en función de las líneas detectadas.
*/
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  /*
    La distancia máxima desde el centro de la imagen hasta
    una esquina, haciendo uso de pitágoras
  */
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  /*
    El tamaño del acumulador se determina multiplicando el radio
    máximo por 2 y por 180 (grados) (2 grados por bin)
  */
  *acc = new int[(int)((round(rMax * 2 *180)))];    
  memset (*acc, 0, sizeof (int) * (int)((round(rMax * 2 *180))));

  /**
   * Centro de la imagen horizontal y verticalmente
  */
  int xCent = w / 2;
  int yCent = h / 2;

  /**
   * Se recorre cada píxel de la imagen. Si el valor del
   * píxel es mayor a 250 (considerado casi blanco),
   * se procede a realizar la transformada de Hough para ese píxel.
  */
  for (int x = 0; x < w; x++) 
    for (int y = 0; y < h; y++)
      {
        int idx = (y * w) + x;

        if (pic[idx] > 250)
          {
            float xCoord = x - xCent;
            float yCoord = y - yCent;
            
            /*
              Se calculan las coordenadas relativas al centro de la imagen y
              se calcula r = x.cos(theta) + y.sin(theta). Se incrementa el acumulador.
            */
            for (int theta = 0; theta < degreeBins; theta++)
              {
                float distance = ( (xCoord) * cos((float)theta * DEG2RAD)) + ((yCoord) * sin((double)theta * DEG2RAD));
                (*acc)[ (int)((round(distance + rMax) * 180)) + theta]++;
              }
          }
      }
}

//*****************************************************************
// DONE usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];
/*
  Transformada de Hough en la GPU. Se realiza en paralelo
  haciendo uso de multiples hilos, donde cada hilo procesa un pixel.
*/
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  /*
    Se calcula el ID basado en el indice de bloque e hilo.
    Si es mayor que el tamaño de la imagen, no se realiza nada.
  */
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  /**
   * Centro de la imagen horizontal y verticalmente
  */
  int xCent = w / 2;
  int yCent = h / 2;

  /*
    Se calculan las coordenadas relativas al centro de la imagen
  */
  int xCoord = gloID % w - xCent;
  int yCoord = (yCent - gloID / w) * -1;

  /**
   * Si el valor del píxel es mayor a 250 (considerado casi blanco),
   * se procede a realizar la transformada de Hough para ese píxel.
  */
  if (pic[gloID] > 250)
    {
      /*
        Se calcula r = x.cos(theta) + y.sin(theta). Se incrementa el acumulador.
      */
      for (int theta = 0; theta < degreeBins; theta++)
        {
          // DONE utilizar memoria constante para senos y cosenos
          float distance = (xCoord * d_Cos[theta]) + (yCoord * d_Sin[theta]);
          /*
            Se hace uso de atomicAdd para manejar actualizaciones concurrentes de múltiples hilos.
            Evitamos race conditions, al hacer que nuestra suma se realice de forma atómica,
            es decir, que no sea afectada por otros hilos que puedan estar realizando la misma
            operación en la misma ubicación de memoria. Así, no se mezclan resultados.
          */
          atomicAdd(&acc[(int)((round(distance + rMax) * 180)) + theta], 1);
        }
    }

}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;
  /*
    En los argumentos, se obtiene la imagen a procesar y el nombre
    de la imagen final.
  */
  const char* inputFileName = argv[1];
  const char* outputFileName = argv[2];
  /*
    Se abre la imagen haciendo uso de CImg
    Se obtiene su ancho u altura
  */
  CImg<unsigned char> image(inputFileName);
  int *cpuht;
  int w = image.width();
  int h = image.height();

  // CPU calculation
  CPU_HoughTran(image.data(), w, h, &cpuht);

  /*
    Se calculan los valores precalculados de coseno y seno
    (pcCos y pcSin) para su uso posterior en la GPU.
  */
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos ((float)i * DEG2RAD);
    pcSin[i] = sin ((float)i * DEG2RAD);
    rad += radInc;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  /*
    Se reserva memoria en la GPU para los arreglos
    d_in (datos de entrada de la imagen) y d_hough (acumulador en la GPU).
  */
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = image.data(); 
  h_hough = (int *) malloc (sizeof (int) * (int)((round(rMax * 2 * 180))));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  /**
   * Se toman los tiempos de ejecucion haciendo uso de un cudaEvent
   */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  /*
    Se configura la cantidad de bloques de acuerdo al tamaño de la imagen
  */
  int blockNum = ceil ((double)w * (double)h / (double)256);
  hipEventRecord(start);
  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);
  hipEventRecord(stop);

  /*
    Se obtienen los resultados y, se comparan contra los del CPU
  */
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }

  /*
    Se muestra el tiempo de ejecucion en pantalla
  */
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Kernel execution time: %f ms\n", milliseconds);

  printf("Done!\n");

  /**
   * Se muestra el resultado en pantalla. Para cada índice en el vector,
   * se revisa si es mayor que el threshold. Si lo es, se guarda la (distancia, angulo).
  */
  CImg<unsigned char> result_image = image;
  unsigned char red[] = {255, 0, 0};
  int threshold = 200; // Ajustable
  float _accu_h = ((sqrt(2.0) * (double)(h>w?h:w)) / 2.0) * 2; 
  std::vector<std::pair<int, int>> indices;
  for(int r=0;r<_accu_h;r++) {
    for(int t=0;t<180;t++) {
      if((int)cpuht[(r*180) + t] >= threshold) {
        indices.push_back(std::make_pair(r, t));
      }
    }
  }
  /*
   * Se calculan las coordenadas de dos puntos que forman una línea detectada en la imagen
  */
  for (const auto& index : indices) {
    int r = index.first;
    int t = index.second;
    int x1, y1, x2, y2;
    x1 = y1 = x2 = y2 = 0;

    if(t >= 45 && t <= 135)
    {
      //y = (r - x cos(t)) / sin(t)
      x1 = 0;
      y1 = ((double)(r-(_accu_h/2)) - ((x1 - (w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (h / 2);
      x2 = w - 0;
      y2 = ((double)(r-(_accu_h/2)) - ((x2 - (w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (h / 2);
    }
    else
    {
      //x = (r - y sin(t)) / cos(t);
      y1 = 0;
      x1 = ((double)(r-(_accu_h/2)) - ((y1 - (h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (w / 2);
      y2 = h - 0;
      x2 = ((double)(r-(_accu_h/2)) - ((y2 - (h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (w / 2);
    }

    result_image.draw_line(x1, y1, x2, y2, red);
  }

  /**
   * Se muestra y se guardan los resultados. 
   */
  result_image.display(); 
  result_image.save(outputFileName); 
  
  /*
    Se liberan los recursos.
  */
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_Cos);
  hipFree(d_Sin);
  hipFree(d_in);
  hipFree(d_hough);
  free(h_hough);
  free(pcCos);
  free(pcSin);

  return 0;
}
