#include "hip/hip_runtime.h"
/*
  ============================================================================
  Author        : G. Barlas
  Version       : 1.0
  Last modified : December 2014
  License       : Released under the GNU GPL 3.0
  Description   :
  To build use  : make
  ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <string.h>
#include "CImg.h"
using namespace cimg_library;

#define M_PI 3.14159265358979323846
#define DEG2RAD (M_PI/180.0f)
const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
const int sharedSize = degreeBins * rBins;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;              //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[(int)((round(rMax * 2 *180)))];                  //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * (int)((round(rMax * 2 *180)))); //init en ceros

  int xCent = w / 2;
  int yCent = h / 2;

  for (int x = 0; x < w; x++) //por cada pixel
    for (int y = 0; y < h; y++) //...
      {
        int idx = (y * w) + x;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            float xCoord = x - xCent;
            float yCoord = y - yCent;
            //  printf("\nCPU: ID: %i; X: %f, Y: %f", idx, xCoord, yCoord);
            for (int theta = 0; theta < degreeBins; theta++)
              {
                // printf("SIN %i: %f\n", theta, (sin((double)theta * DEG2RAD)));
                float distance = ( (xCoord) * cos((float)theta * DEG2RAD)) + ((yCoord) * sin((double)theta * DEG2RAD));
                /*if (theta == 90)
                  printf("\nCPU: ID: %i THETA: %i, DISTANCE: %f", idx, theta, distance);
                */(*acc)[ (int)((round(distance + rMax) * 180)) + theta]++; //+1 para este radio distance y este theta
              }
          }
      }
}

//*****************************************************************
// DONE usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTranShared (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  __shared__ int localAcc[sharedSize];                      //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000

  int localID = threadIdx.x;
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadIdx.x; i < sharedSize; i += blockDim.x) 
    localAcc[i] = 0;

  if (gloID > w * h) return;                                // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = (yCent - gloID / w) * -1;

  __syncthreads();
  if (pic[gloID] > 0)
  {
    for (int theta = 0; theta < degreeBins; theta++)
    {
      float r = xCoord * d_Cos[theta] + yCoord * d_Sin[theta];
      float distance = (xCoord * d_Cos[theta]) + (yCoord * d_Sin[theta]);
      atomicAdd(&localAcc[(int)((round(distance + rMax) * 180)) + theta], 1); //+1 para este radio distance y este theta
    }
  }

  __syncthreads();
  for (int i = threadIdx.x; i < sharedSize; i += blockDim.x) 
  {
    atomicAdd(acc, localAcc[i]);
  }
}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  const char* inputFileName = argv[1];
  const char* outputFileName = argv[2];

  CImg<unsigned char> image(inputFileName);

  int *cpuht;
  int w = image.width();
  int h = image.height();

  // CPU calculation
  CPU_HoughTran(image.data(), w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos ((float)i * DEG2RAD);
    pcSin[i] = sin ((float)i * DEG2RAD);
    // printf("Sin of %i: %f\n", i, pcSin[i]);
    rad += radInc;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = image.data(); // h_in contiene los pixeles de la imagen
  h_hough = (int *) malloc (sizeof (int) * (int)((round(rMax * 2 * 180))));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int blockNum = ceil ((double)w * (double)h / (double)256);
  hipEventRecord(start);
  GPU_HoughTranShared <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);
  hipEventRecord(stop);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Kernel execution time: %f ms\n", milliseconds);

  printf("Done!\n");

  // Guardando como png
  CImg<unsigned char> result_image = image;
  unsigned char red[] = {255, 0, 0};

  int threshold = 200;  // AJUSTAR SEGUN LO NECESARIO
  // Guardando los valores arriba del threshold

  float _accu_h = ((sqrt(2.0) * (double)(h>w?h:w)) / 2.0) * 2; 
  std::vector<std::pair<int, int>> indices;
  for(int r=0;r<_accu_h;r++)
  {
    for(int t=0;t<180;t++)
    {
      if((int)cpuht[(r*180) + t] >= threshold)
      {
        indices.push_back(std::make_pair(r, t));
      }
    }
  }

  for (const auto& index : indices)
  {
    int r = index.first;
    int t = index.second;
    int x1, y1, x2, y2;
    x1 = y1 = x2 = y2 = 0;

    if(t >= 45 && t <= 135)
    {
      //y = (r - x cos(t)) / sin(t)
      x1 = 0;
      y1 = ((double)(r-(_accu_h/2)) - ((x1 - (w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (h / 2);
      x2 = w - 0;
      y2 = ((double)(r-(_accu_h/2)) - ((x2 - (w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (h / 2);
    }
    else
    {
      //x = (r - y sin(t)) / cos(t);
      y1 = 0;
      x1 = ((double)(r-(_accu_h/2)) - ((y1 - (h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (w / 2);
      y2 = h - 0;
      x2 = ((double)(r-(_accu_h/2)) - ((y2 - (h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (w / 2);
    }

    result_image.draw_line(x1, y1, x2, y2, red);
  }

  // Display or save the resulting image with detected lines
  result_image.display();  // Display the image using CImg's built-in display function
  result_image.save(outputFileName);  // Save the image with detected lines to a file

  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_Cos);
  hipFree(d_Sin);
  hipFree(d_in);
  hipFree(d_hough);
  free(h_hough);
  free(pcCos);
  free(pcSin);

  return 0;
}
